#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling approach.
 * It has been written for clarity of exposition to illustrate various CUDA programming
 * principles, not with the goal of providing the most performant generic kernel for matrix multiplication.
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set on CPU
//! C = A * B
//! @param C          reference data, computed but preallocated
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param hA         height of matrix A
//! @param wA         width of matrix A
//! @param hB         height of matrix B
//! @param wB         width of matrix B
////////////////////////////////////////////////////////////////////////////////
void ComputeGold(float *C, const float *A, const float *B, uint32_t hA, uint32_t wA, uint32_t hB, uint32_t wB) {
  assert(wA == hB);
  for (uint32_t i=0; i<hA; i++) {
    for (uint32_t j=0; j<wB; j++) {
      double sum = 0;
      for (int k=0; k<wA; k++) {
        //sum += A[i][k] * B[k][j]
        sum += A[i*wA+k] * B[k*wB+j];
      }
      //C[i][j] = sum;
      C[i*wB+j] = sum;
    }
  }
}


////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set on GPU
//! C = A * B
//! @param C          reference data, computed but preallocated
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param hA         height of matrix A
//! @param wA         width of matrix A
//! @param hB         height of matrix B
//! @param wB         width of matrix B
////////////////////////////////////////////////////////////////////////////////
__global__ void MatMulKernel_naive(float *C, const float *A, const float *B, uint32_t hA, uint32_t wA, uint32_t hB, uint32_t wB) {
  assert(wA == hB);
  float sum = 0;
  uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t column = blockIdx.x * blockDim.x + threadIdx.x;
  for (uint32_t k=0; k<wA; k++) {
    //sum += A[i][k] * B[k][j];
    sum += A[row * wA + k] * B[k * wB + column];
  }
  //C[i][j] = sum;
  C[wB * row + column] = sum;
}


/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> __global__ void MatrixMulCUDA(float *C, float *A,
    float *B, int wA,
    int wB) {
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int aBegin = wA * BLOCK_SIZE * by;
  int aEnd = aBegin + wA - 1;
  int aStep = BLOCK_SIZE;
  int bBegin = BLOCK_SIZE * bx;
  int bStep = BLOCK_SIZE * wB;
  float Csub = 0;
  for (int a=aBegin, b=bBegin;
        a<=aEnd;
        a+=aStep, b+=bStep) {
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // load the matrices from device memory to shared memory
    As[ty][tx] = A[a + wA*ty +tx];
    Bs[ty][tx] = B[b + wB*ty + tx];
    // make sure the sub-matrices are loaded
    __syncthreads();

    // multiply two matrices together
    //#pragma unroll
    for (int k=0; k<BLOCK_SIZE; k++) {
      Csub += As[ty][k] * Bs[k][tx];
    }
    // make sure that preceding computation is done before loading two new sub-matrices of A and B the next time
    __syncthreads();
  }
  // write computation result to device memory
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB*ty + tx] = Csub;
}


void ConstantInit(float *data, int size, float val) {
  for (int i = 0; i < size; ++i) {
    data[i] = val;
  }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv,
                   int block_size, const dim3 &dimsA,
                   const dim3 &dimsB) {
/*
CUDA programm ABC:
    1. prepare host and device memory
    2. h2d: copy vars from host memory to device memory
    3. perform computation on GPU
    4. d2h: copy computation result from device memory to host memory
    5. destroy device and host memory
*/

  // 1. prepare host and device memory
  printf("1. prepare host and device memory\n");
  uint32_t size_A = dimsA.x * dimsA.y;
  uint32_t mem_size_A = size_A * sizeof(float);
  uint32_t size_B = dimsB.x * dimsB.y;
  uint32_t mem_size_B = size_B * sizeof(float);
  dim3 dimsC(dimsB.x, dimsA.y, 1);
  uint32_t size_C = dimsC.x * dimsC.y;
  uint32_t mem_size_C = size_C * sizeof(float);

  /*
  template<class T>
  static __inline__ __host__ hipError_t hipHostMalloc(
    T            **ptr,
    size_t         size,
    unsigned int   flags = 0
  )
  */
  float *h_A;
  checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
  float *h_B;
  checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
  float *h_C;
  checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));
  const float valB = 0.01f;
  ConstantInit(h_A, size_A, 1.0f);
  ConstantInit(h_B, size_B, valB);

  float *d_A, *d_B, *d_C;
  checkCudaErrors(hipMalloc((void**)&d_A, mem_size_A));
  checkCudaErrors(hipMalloc((void**)&d_B, mem_size_B));
  checkCudaErrors(hipMalloc((void**)&d_C, mem_size_C));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  // 2. h2d: copy vars from host memory to device memory
  printf("2. h2d: copy vars from host memory to device memory\n");
  /*
  hipError_t hipMemcpyAsync(void *dst, const void *src, size_t count, hipMemcpyKind kind, hipStream_t stream = (hipStream_t)0)
  */
  checkCudaErrors(hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
  checkCudaErrors(hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));
  
  // 3. perform computation on GPU
  printf("3. perform computation on GPU\n");
  dim3 threads(block_size, block_size);
  dim3 grid(dimsB.x/threads.x, dimsA.y/threads.y);

  // warmup
  if (block_size == 16) {
    MatrixMulCUDA<16><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
  } else {
    MatrixMulCUDA<32><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
  }
  printf("warmup done\n");
  checkCudaErrors(hipStreamSynchronize(stream));

  // create CUDA events that we'll use for timing
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  // record the start event
  checkCudaErrors(hipEventRecord(start, stream));
  // profile kernel performance
  int n_iter = 300;
  for (int i=0; i<n_iter; i++) {
    if (block_size == 16) {
      MatrixMulCUDA<16><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    } else {
      MatrixMulCUDA<32><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
  }
  // record the stop event
  checkCudaErrors(hipEventRecord(stop, stream));
  // wait for the stop event to complete
  checkCudaErrors(hipEventSynchronize(stop));

  // calculate average latency
  float ms_total = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&ms_total, start, stop));
  float ms_per_mm = ms_total/n_iter;
  // to compute C[i][j] we have to perform multiplication wA times and addition wA-1 times (about 2*wA times ops)
  // so the total ops would be (2*wA * hA*wB) times
  double flops_per_mm = 2.0 * dimsA.x * dimsA.y * dimsB.x;
  double giga_flops = (flops_per_mm * 1e-9f) / (ms_per_mm/1000.0f);
  printf(
      "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,"
      " WorkgroupSize= %u threads/block\n",
      giga_flops, ms_per_mm, flops_per_mm, threads.x * threads.y);

  // 4. d2h: copy computation result from device memory to host memory
  printf("4. d2h: copy computation result from device memory to host memory\n");
  checkCudaErrors(hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  // 4.1 verify result
  bool correct = true;

  /*
  // 4.1.1 verify result with CPU computation result
  float *ref_C;
  checkCudaErrors(hipHostMalloc((void**)&ref_C, mem_size_C));
  ComputeGold(ref_C, h_A, h_B, dimsA.y, dimsA.x, dimsB.y, dimsB.x);
  double eps = 1e-5;
  for (uint32_t i=0; i<size_C; i++) {
    if (fabs(h_C[i] - ref_C[i]) > eps) {
      printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
              i, h_C[i], ref_C[i], eps);
      correct = false;
    }
  }
  checkCudaErrors(hipHostFree(ref_C));
  */

  // 4.1.2 verify result with GPU computation result
  float *ref_C, *ref_d_C;
  checkCudaErrors(hipHostMalloc((void**)&ref_C, mem_size_C));
  checkCudaErrors(hipMalloc((void**)&ref_d_C, mem_size_C));
  MatMulKernel_naive<<<grid, threads, 0, stream>>>(ref_d_C, h_A, h_B, dimsA.y, dimsA.x, dimsB.y, dimsB.x);
  checkCudaErrors(hipMemcpyAsync(ref_C, ref_d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipStreamSynchronize(stream));
  double eps = 1e-6;
  for (uint32_t i=0; i<size_C; i++) {
    if (fabs(h_C[i] - ref_C[i]) > eps) {
      printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
              i, h_C[i], ref_C[i], eps);
      correct = false;
    }
  }
  checkCudaErrors(hipFree(ref_d_C));
  checkCudaErrors(hipHostFree(ref_C));

  /*
  // test relative error by the formula: |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
  double eps = 1e-6;
  for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
    double abs_err = fabs(h_C[i] - (dimsA.x * valB));
    double dot_length = dimsA.x;
    double abs_val = fabs(h_C[i]);
    double rel_err = abs_err / abs_val / dot_length;
    if (rel_err > eps) {
      printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
             i, h_C[i], dimsA.x * valB, eps);
      correct = false;
    }
  }
  */

  printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

  // 5. destroy device and host memory
  printf("5. destroy device and host memory\n");
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));
  checkCudaErrors(hipHostFree(h_A));
  checkCudaErrors(hipHostFree(h_B));
  checkCudaErrors(hipHostFree(h_C));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  printf(
      "\nNOTE: The CUDA Samples are not meant for performance "
      "measurements. Results may vary when GPU Boost is enabled.\n");

  return correct ? EXIT_SUCCESS : EXIT_FAILURE;
}


/**
 * Program main
 */
int main(int argc, char **argv) {
  printf("[Matrix Multiply Using CUDA] - Starting...\n");

  if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
      checkCmdLineFlag(argc, (const char **)argv, "?")) {
    printf("Usage -device=n (n >= 0 for deviceID)\n");
    printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
    printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
    printf("  Note: Outer matrix dimensions of A & B matrices" \
           " must be equal.\n");

    exit(EXIT_SUCCESS);
  }

  // This will pick the best possible CUDA capable device, otherwise
  // override the device ID based on input provided at the command line
  int dev = findCudaDevice(argc, (const char **)argv);

  int block_size = 32;

  dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
  dim3 dimsB(5 * 4 * block_size, 5 * 2 * block_size, 1);

  // width of Matrix A
  if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
    dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
  }

  // height of Matrix A
  if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
    dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
  }

  // width of Matrix B
  if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
    dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
  }

  // height of Matrix B
  if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
    dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
  }

  if (dimsA.x != dimsB.y) {
    printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
           dimsA.x, dimsB.y);
    exit(EXIT_FAILURE);
  }

  printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
         dimsB.x, dimsB.y);

  checkCudaErrors(hipProfilerStart());
  int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB);
  checkCudaErrors(hipProfilerStop());

  exit(matrix_result);
}
