#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// a note: https://kharshit.github.io/blog/2024/06/07/matrix-multiplication-cuda

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set on CPU
//! C = A * B
//! @param C          reference data, computed but preallocated
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param hA         height of matrix A
//! @param wA         width of matrix A
//! @param hB         height of matrix B
//! @param wB         width of matrix B
////////////////////////////////////////////////////////////////////////////////
void ComputeGold(float *C, const float *A, const float *B, uint32_t hA, uint32_t wA, uint32_t hB, uint32_t wB) {
  assert(wA == hB);
  for (uint32_t i=0; i<hA; ++i) {
    for (uint32_t j=0; j<wB; ++j) {
      double sum = 0;
      for (int k=0; k<wA; k++) {
        //sum += A[i][k] * B[k][j]
        sum += A[i*wA+k] * B[k*wB+j];
      }
      //C[i][j] = sum;
      C[i*wB+j] = sum;
    }
  }
}


////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set on GPU
//! C = A * B
//! @param C          reference data, computed but preallocated
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param hA         height of matrix A
//! @param wA         width of matrix A
//! @param hB         height of matrix B
//! @param wB         width of matrix B
////////////////////////////////////////////////////////////////////////////////
__global__ void MatMulKernel_naive(float *C, const float *A, const float *B, uint32_t hA, uint32_t wA, uint32_t hB, uint32_t wB) {
  assert(wA == hB);
  uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t column = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < hA && column < wB) {
    float sum = 0;
    for (uint32_t k=0; k<wA; ++k) {
      //sum += A[i][k] * B[k][j];
      sum += A[row * wA + k] * B[k * wB + column];
    }
    //C[i][j] = sum;
    C[row * wB + column] = sum;
  }
}

template <int TILE_SIZE> __global__ void MatrixMulCUDAWithTile(float *C, float *A,
    float *B, int hA, int wA,
    int hB, int wB) {
  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  // element `C[row][column]` to be calculated
  uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t column = blockIdx.x * blockDim.x + threadIdx.x;

  // shared memory arrays used to store sub-matrix
  // shared by threads in a thread block
  __shared__ float As[TILE_SIZE][TILE_SIZE];
  __shared__ float Bs[TILE_SIZE][TILE_SIZE];

  // `Csub` is private to each thread, and may store on registers
  float Csub = 0;
  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  // and we still iterate for `wA` time to calculate `Csub`
  for (int m=0; m<(wA+TILE_SIZE-1)/TILE_SIZE; m++) {
    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    if (row<hA && (m * TILE_SIZE + tx)<wA) {
      As[ty][tx] = A[row * wA + (m * TILE_SIZE + tx)];
    } else {
      As[ty][tx] = 0;
    }
    if ((m * TILE_SIZE + ty)<hB && column<wB) {
      Bs[ty][tx] = B[(m * TILE_SIZE + ty) * wB + column];
    } else {
      Bs[ty][tx] = 0;
    }
    // Synchronize to make sure the matrices are loaded
    __syncthreads();
    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
    #pragma unroll
    for (int k=0; k<TILE_SIZE; k++) {
      Csub += As[ty][k] * Bs[k][tx];
    }
    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }
  // write final result to device memory
  C[row * wB + column] = Csub;
}

void ConstantInit(float *data, int size, float val) {
  for (int i = 0; i < size; ++i) {
    data[i] = val;
  }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv,
                   int block_size, const dim3 &dimsA,
                   const dim3 &dimsB) {
/*
CUDA programm ABC:
    1. prepare host and device memory
    2. h2d: copy vars from host memory to device memory
    3. perform computation on GPU
    4. d2h: copy computation result from device memory to host memory
    5. destroy device and host memory
*/

  // 1. prepare host and device memory
  printf("1. prepare host and device memory\n");
  uint32_t size_A = dimsA.x * dimsA.y;
  uint32_t mem_size_A = size_A * sizeof(float);
  uint32_t size_B = dimsB.x * dimsB.y;
  uint32_t mem_size_B = size_B * sizeof(float);
  dim3 dimsC(dimsB.x, dimsA.y, 1);
  uint32_t size_C = dimsC.x * dimsC.y;
  uint32_t mem_size_C = size_C * sizeof(float);

  float *h_A, *h_B, *h_C;
  checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
  checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
  checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));
  const float valB = 0.01f;
  ConstantInit(h_A, size_A, 1.0f);
  ConstantInit(h_B, size_B, valB);

  float *d_A, *d_B, *d_C;
  checkCudaErrors(hipHostGetDevicePointer((void**)&d_A, (void*)h_A, 0));
  checkCudaErrors(hipHostGetDevicePointer((void**)&d_B, (void*)h_B, 0));
  checkCudaErrors(hipHostGetDevicePointer((void**)&d_C, (void*)h_C, 0));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  // 2. h2d: copy vars from host memory to device memory
  printf("2. h2d: copy vars from host memory to device memory\n");
  
  // 3. perform computation on GPU
  printf("3. perform computation on GPU\n");
  dim3 threads(block_size, block_size);
  dim3 grid(dimsB.x/threads.x, dimsA.y/threads.y);

  // warmup
  if (block_size == 16) {
    MatrixMulCUDAWithTile<16><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.y, dimsA.x, dimsB.y, dimsB.x);
  } else {
    MatrixMulCUDAWithTile<32><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.y, dimsA.x, dimsB.y, dimsB.x);
  }
  printf("warmup done\n");
  checkCudaErrors(hipStreamSynchronize(stream));

  // create CUDA events that we'll use for timing
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  // record the start event
  checkCudaErrors(hipEventRecord(start, stream));
  // profile kernel performance
  int n_iter = 300;
  for (int i=0; i<n_iter; ++i) {
    if (block_size == 16) {
      MatrixMulCUDAWithTile<16><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.y, dimsA.x, dimsB.y, dimsB.x);
    } else {
      MatrixMulCUDAWithTile<32><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.y, dimsA.x, dimsB.y, dimsB.x);
    }
  }
  // record the stop event
  checkCudaErrors(hipEventRecord(stop, stream));
  // wait for the stop event to complete
  checkCudaErrors(hipEventSynchronize(stop));

  // calculate average latency
  float ms_total = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&ms_total, start, stop));
  float ms_per_mm = ms_total/n_iter;
  // to compute C[i][j] we have to perform wA times multiplication and wA times addition
  // so the total ops would be (2*wA * hA * wB) times
  double flops_per_mm = 2.0 * dimsA.x * dimsA.y * dimsB.x;
  double giga_flops = (flops_per_mm * 1e-9f) / (ms_per_mm/1000.0f);
  printf(
      "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,"
      " WorkgroupSize= %u threads/block\n",
      giga_flops, ms_per_mm, flops_per_mm, threads.x * threads.y);

  // 4. d2h: copy computation result from device memory to host memory
  printf("4. d2h: copy computation result from device memory to host memory\n");

  // 4.1 verify result
  bool correct = true;
  // 4.1.2 verify result with GPU computation result
  float *ref_C, *ref_d_C;
  checkCudaErrors(hipHostMalloc((void**)&ref_C, mem_size_C));
  checkCudaErrors(hipHostGetDevicePointer((void**)&ref_d_C, (void*)ref_C, 0));
  MatMulKernel_naive<<<grid, threads, 0, stream>>>(ref_d_C, d_A, d_B, dimsA.y, dimsA.x, dimsB.y, dimsB.x);
  checkCudaErrors(hipStreamSynchronize(stream));
  double eps = 1e-6;
  for (uint32_t i=0; i<size_C; ++i) {
    if (fabs(h_C[i] - ref_C[i]) > eps) {
      printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
              i, h_C[i], ref_C[i], eps);
      correct = false;
    }
  }
  checkCudaErrors(hipHostFree(ref_C));

  printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

  // 5. destroy device and host memory
  printf("5. destroy device and host memory\n");
  checkCudaErrors(hipHostFree(h_A));
  checkCudaErrors(hipHostFree(h_B));
  checkCudaErrors(hipHostFree(h_C));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  printf(
      "\nNOTE: The CUDA Samples are not meant for performance "
      "measurements. Results may vary when GPU Boost is enabled.\n");

  return correct ? EXIT_SUCCESS : EXIT_FAILURE;
}


/**
 * Program main
 */
int main(int argc, char **argv) {
  printf("[Matrix Multiply Using CUDA] - Starting...\n");

  if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
      checkCmdLineFlag(argc, (const char **)argv, "?")) {
    printf("Usage -device=n (n >= 0 for deviceID)\n");
    printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
    printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
    printf("  Note: Outer matrix dimensions of A & B matrices" \
           " must be equal.\n");

    exit(EXIT_SUCCESS);
  }

  // This will pick the best possible CUDA capable device, otherwise
  // override the device ID based on input provided at the command line
  int idev = findCudaDevice(argc, (const char **)argv);

  int block_size = 32;

  dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
  dim3 dimsB(5 * 4 * block_size, 5 * 2 * block_size, 1);

  // width of Matrix A
  if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
    dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
  }

  // height of Matrix A
  if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
    dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
  }

  // width of Matrix B
  if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
    dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
  }

  // height of Matrix B
  if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
    dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
  }

  if (dimsA.x != dimsB.y) {
    printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
           dimsA.x, dimsB.y);
    exit(EXIT_FAILURE);
  }

  printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
         dimsB.x, dimsB.y);

  /*
  Set device to be used for GPU executions
  Sets device as the current device for the calling host thread. Valid device id's are 0 to (::hipGetDeviceCount() - 1).
  */
  checkCudaErrors(hipSetDevice(idev));
  /* To be able to retrieve the device pointer to any mapped page-locked memory, page-locked memory mapping must be enabled by calling hipSetDeviceFlags() with the hipDeviceMapHost flag before any other CUDA call is performed. Otherwise, hipHostGetDevicePointer() will return an error. */
  checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));

  checkCudaErrors(hipProfilerStart());
  int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB);
  checkCudaErrors(hipProfilerStop());

  exit(matrix_result);
}

/* oops, ten times slower than case where we don't use mapped memory
root@di-20241115115906-kfh5w:~/code/cuda-samples/Samples/0_Introduction/matrixMul# ./matrixMul 
[Matrix Multiply Using CUDA] - Starting...
GPU Device 0: "Ada" with compute capability 8.9

MatrixA(320,320), MatrixB(640,320)
1. prepare host and device memory
2. h2d: copy vars from host memory to device memory
3. perform computation on GPU
warmup done
Performance= 208.38 GFlop/s, Time= 0.629 msec, Size= 131072000 Ops, WorkgroupSize= 1024 threads/block
4. d2h: copy computation result from device memory to host memory
Result = PASS
5. destroy device and host memory

NOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.
*/